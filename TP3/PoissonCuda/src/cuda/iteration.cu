#include "hip/hip_runtime.h"
#include "dim.cuh"
#include "cuda_check.cuh"
#include "user.cuh"

#include "timer.hxx"
#include "iteration.hxx"



// A completer : definition du noyau

__global__ void test_kernel(){

    int nx = d_n[0];
    int ny = d_n[1];
    int nz = d_n[2];
    double dx = d_dx[0];
    double dy = d_dx[1];
    double dz = d_dx[2];

    double xmin = d_xmin[0];
    double ymin = d_xmin[1];
    double zmin = d_xmin[2];

    printf("nx = %d ny = %d nz = %d\ndx = %f dy = %f dz = %f\nxmin = %f ymin = %f zmin = %f\n",nx,ny,nz,dx,dy,dz,xmin,ymin,zmin);

    double lam_x = d_lambda[0];
    double lam_y = d_lambda[1];
    double lam_z = d_lambda[2];

    printf("lam_x = %f lam_x = %f lam_z = %f",lam_x,lam_y,lam_z);
    
}
__global__ void iteration_kernel(
    double*u, double *v, const double dt,
    const int imin, const int imax, 
    const int jmin, const int jmax,
    const int kmin, const int kmax
    ) {

    int nx = d_n[0];
    int ny = d_n[1];
    //int nz = d_n[2];

    double dx = d_dx[0];
    double dy = d_dx[1];
    double dz = d_dx[2];

    double xmin = d_xmin[0];
    double ymin = d_xmin[1];
    double zmin = d_xmin[2];

    int i = blockIdx.x * blockDim.x + threadIdx.x + imin;
    int j = blockIdx.y * blockDim.y + threadIdx.y + jmin;
    int k = blockIdx.z * blockDim.z + threadIdx.z + kmin;

    int n1 = nx;
    int n2 = ny * nx;

    if (i > imax || j > jmax || k > kmax)
        return;


    double lam_x = d_lambda[0];
    double lam_y = d_lambda[1];
    double lam_z = d_lambda[2];

    


    double du1 = (-2 * u[i + n1*j + n2*k] + u[i+1 + n1*j + n2*k] + u[i-1 + n1*j + n2*k]) * lam_x
            + (-2 * u[i + n1*j + n2*k] + u[i + n1*(j+1) + n2*k] + u[i + n1*(j-1) + n2*k]) * lam_y
            + (-2 * u[i + n1*j + n2*k] + u[i + n1*j + n2*(k+1)] + u[i + n1*j + n2*(k-1)]) * lam_z;

    double x = xmin + i * dx;
    double y = ymin + j * dy;
    double z = zmin + k * dz;
    double du2 = force(x, y, z);

    double du = dt * (du1 + du2);
    v[i + n1*j + n2*k] = u[i + n1*j + n2*k] + du;

}

__global__ void iteration_kernel_coarse(
    double*u, double *v, const double dt,
    const int imin, const int imax, 
    const int jmin, const int jmax,
    const int kmin, const int kmax,
    const int sx, const int sy, const int sz
    ) {

    int nx = d_n[0];
    int ny = d_n[1];
    //int nz = d_n[2];

    double dx = d_dx[0];
    double dy = d_dx[1];
    double dz = d_dx[2];

    double xmin = d_xmin[0];
    double ymin = d_xmin[1];
    double zmin = d_xmin[2];

    int i = sx *(blockIdx.x * blockDim.x + threadIdx.x) + imin;
    int j = sy *(blockIdx.y * blockDim.y + threadIdx.y) + jmin;
    int k = sz *(blockIdx.z * blockDim.z + threadIdx.z) + kmin;

    int n1 = nx;
    int n2 = ny * nx;

    if (i > imax || j > jmax || k > kmax)
        return;


    double lam_x = d_lambda[0];
    double lam_y = d_lambda[1];
    double lam_z = d_lambda[2];
    
    int ii_max = min(i + sx, imax + 1);
    int jj_max = min(j + sy, jmax + 1);
    int kk_max = min(k + sz, kmax + 1);

    
    for(int kk = k; kk < kk_max; kk++){
        for(int jj = j; jj < jj_max; jj++){
            for(int ii = i; ii < ii_max; ii++){
                double du1 = (-2 * u[ii + n1*jj + n2*kk] + u[ii+1 + n1*jj + n2*kk] + u[ii-1 + n1*jj + n2*kk]) * lam_x
            + (-2 * u[ii + n1*jj + n2*kk] + u[ii + n1*(jj+1) + n2*kk] + u[ii + n1*(jj-1) + n2*kk]) * lam_y
            + (-2 * u[ii + n1*jj + n2*kk] + u[ii + n1*jj + n2*(kk+1)] + u[ii + n1*jj + n2*(kk-1)]) * lam_z;

            double x = xmin + ii * dx;
            double y = ymin + jj * dy;
            double z = zmin + kk * dz;
            double du2 = force(x, y, z);

            double du = dt * (du1 + du2);
            v[ii + n1*jj + n2*kk] = u[ii + n1*jj + n2*kk] + du;
            }
        }
    }

    

}


int get_xyz_size_opt(const unsigned int blockSize, const unsigned int ijk_size, const unsigned int N){
    return ceil((float)ijk_size/(float)(blockSize*N));
}

dim3 get_gridSize(const dim3& blockSize, const dim3& ijk_size, const dim3& xyz_size = dim3(1,1,1)){
    int bx = blockSize.x;
    int by = blockSize.y;
    int bz = blockSize.z;


    int si = ijk_size.x;
    int sj = ijk_size.y;
    int sk = ijk_size.z;
    

    int sx = max(1,xyz_size.x);
    int sy = max(1,xyz_size.y);
    int sz = max(1,xyz_size.z);


    dim3 gridSize(ceil((float)(si) / (float)(bx * sx)),
              ceil((float)(sj) / (float)(by * sy)),
              ceil((float)(sk) / (float)(bz * sz)));

    //printf("s_ijk / (blocSize * s_xyz)\n%.3f %.3f %.3f\n",(float)(si) / (float)(bx * sx),(float)(sj) / (float)(by * sy),(float)(sk) / (float)(bz * sz));
    //printf("gridSize\n%d %d %d\n",gridSize.x,gridSize.y,gridSize.z);

    return gridSize;
}



void test_dim_cu(){
    test_kernel<<<1,1>>>();
}

void test_xyz_size_opt(const unsigned int ni, const unsigned int blockSize,bool simple=false){
    int blockSize_max = ceil((float)(ni) / (float)(blockSize));
    int rec = 0;
    for(int n=1; n<= blockSize_max; n++){
        int sx = get_xyz_size_opt(blockSize,ni,n);
        if(sx==rec&&simple){
            continue;
        }else{
            rec = sx;
        }
        float gsf = (float)(ni) / (float)(blockSize * sx);
        int gs = ceil(gsf);
        float eff = 100.0 * gsf /(float)gs;
        printf("n = %d, sx = %d, gridSize = %d efficiency = %.1f %%\n",n,sx,gs,eff);
    }
}

void test_block_size(const unsigned int ni = 399, unsigned int blockSize_max = 256){
    for(int th = 1; th <= blockSize_max; th++){
        printf("_______________________________\n");
        printf("blockSize = %d\n",th);
        test_xyz_size_opt(ni,th,true);
    }
}
void iteration(
    Values & v, Values & u, double dt, int n[3],
    int imin, int imax, 
    int jmin, int jmax,
    int kmin, int kmax)
{
// A completer : appel du noyau

    dim3 ijk_size(imax - imin + 1,jmax - jmin + 1,kmax - kmin + 1);

    const bool is_coarse = false;

    double* d_u = u.dataGPU();
    double* d_v = v.dataGPU();

    if(is_coarse){
        unsigned int bx = 57;
        unsigned int by = 7;
        unsigned int bz = 1;
        dim3 blockSize(bx,by,bz); // block size can be adjusted

        dim3 xyz_size_opt(
        get_xyz_size_opt(bx,ijk_size.x,1),
        get_xyz_size_opt(by,ijk_size.y,19),
        get_xyz_size_opt(bz,ijk_size.z,400));

        dim3 gridSize = get_gridSize(blockSize,ijk_size,xyz_size_opt);
        int sx = xyz_size_opt.x;
        int sy = xyz_size_opt.y;
        int sz = xyz_size_opt.z;
        iteration_kernel_coarse<<<gridSize, blockSize>>>(
        d_u, d_v, dt, imin, imax, jmin, jmax, kmin, kmax, sx,sy,sz);
    }else{
        unsigned int bx = 8;
        unsigned int by = 8;
        unsigned int bz = 8;
        dim3 blockSize(bx,by,bz); // block size can be adjusted
        dim3 gridSize = get_gridSize(blockSize,ijk_size);
        iteration_kernel<<<gridSize, blockSize>>>(
        d_u, d_v, dt, imin, imax, jmin, jmax, kmin, kmax);
    }
    hipDeviceSynchronize(); // synchronize to ensure kernel execution finishes

}
